
#include <hip/hip_runtime.h>
__global__ void wave1Dmac1(double * f_tmp1, double * f_in,
				double u, double dt, double dx,
				int N){
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if(tid<N){
    int x_p = tid+1;
    if(x_p == N) x_p = 0;
    
    double f_tmp = f_in[tid];
    f_tmp1[tid]= f_tmp - u*(dt/dx)*(f_in[x_p] - f_tmp);

  }
}
