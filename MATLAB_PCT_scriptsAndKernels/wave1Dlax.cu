
#include <hip/hip_runtime.h>
__global__ void wave1Dlax(double * f_next, double * f, double u, 
			  double dt, double dx, int N){
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if(tid<N){
    int x_p = tid+1;
    if(x_p ==N)
      x_p = 0;
    int x_m = tid-1;
    if(x_m<0)
      x_m = N-1;

    
    double f_p = f[x_p];
    double f_m = f[x_m];

    f_next[tid]=0.5*(f_p + f_m) - (u*dt/(2.*dx))*(f_p - f_m);

  }
}
