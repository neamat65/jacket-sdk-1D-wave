
#include <hip/hip_runtime.h>
__global__ void wave1Dmac2(double * f_next, double * f_tmp1,
				double * f_in, double u, double dt,
				double dx, int N){
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if(tid<N){
    int x_m = tid-1;
    if(x_m <0) x_m = N-1;

    double ft1_tmp = f_tmp1[tid];
    f_next[tid]=0.5*(f_in[tid]+ft1_tmp - u*(dt/dx)*(ft1_tmp-f_tmp1[x_m]));

  }
}
