
#include <hip/hip_runtime.h>
__global__ void wave1Drusanov3(double * f_next,double * f_tmp, 
				double * f_in, double nu,
				double omega, int N){
  int tid=threadIdx.x+blockIdx.x*blockDim.x;
  if(tid<N){
    int x_2m=tid-2;
    if(x_2m<0) x_2m+=N;
    int x_m = tid-1;
    if(x_m<0) x_m+=N;

    int x_p = tid+1;
    if(x_p>(N-1)) x_p-=N;

    int x_2p = tid+2;
    if(x_2p>(N-1)) x_2p-=N;

    double f_2m = f_in[x_2m];
    double f_m = f_in[x_m];
    double f = f_in[tid];
    double f_p = f_in[x_p];
    double f_2p = f_in[x_2p];

    f_next[tid]=f-(nu/24.)*(-2.*f_2p+7.*f_p - 7.*f_m+2.*f_2m)
      -(3.*nu/8.)*(f_tmp[x_p]-f_tmp[x_m])
      -(omega/24.)*(f_2p - 4.*f_p + 6.*f - 4.*f_m + f_2m);


  }
}
