
#include <hip/hip_runtime.h>
__global__ void wave1Drusanov2(double * f_tmp,double * f_nm, 
				double * f_in, double nu, int N){
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if(tid<N){
    int x_m = tid-1;
    if(x_m<0) x_m = (N-1);
    f_tmp[tid]=f_in[tid]-(2.*nu/3.)*(f_nm[tid]-f_nm[x_m]);

  }
}
