
#include <hip/hip_runtime.h>
__global__ void wave1Drusanov1(double * f_nm, double * f_in, 
				double nu,int N){

  int tid=threadIdx.x+blockIdx.x*blockDim.x;
  if(tid<N){
    int x_p = tid+1;
    if(x_p==N) x_p=0;

    double fp = f_in[x_p];
    double f = f_in[tid];
    f_nm[tid]=0.5*(fp+f)-(nu/3.)*(fp-f);

  }
}
